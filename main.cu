/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>

#include "support.h"
#include "kernel.cu"

int main(int argc, char* argv[])
{
    Timer timer;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

	float *in_h, *out2_h, *out3_h, *out_h;
	float *in_d, *out2_d, *out3_d, *out_d;
	unsigned num_elements;
	hipError_t cuda_ret;

	/* Allocate and initialize input vector */
    if(argc == 1) {
        num_elements = 1000000;
    } else if(argc == 2) {
        num_elements = atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./prefix-scan        # Input of size 1,000,000 is used"
           "\n    Usage: ./prefix-scan <m>    # Input of size m is used"
           "\n");
        exit(0);
    }
    initVector(&in_h, num_elements);


	/* Allocate and initialize output vector */
	out2_h = (float*)calloc(num_elements, sizeof(float));
	if(out2_h == NULL) FATAL("Unable to allocate host");

	out3_h = (float*)calloc(SECTION_SIZE, sizeof(float));
	if(out3_h == NULL) FATAL("Unable to allocate host");

	out_h = (float*)calloc(num_elements, sizeof(float));
	if(out_h == NULL) FATAL("Unable to allocate host");

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Input size = %u\n", num_elements);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

	cuda_ret = hipMalloc((void**)&in_d, num_elements*sizeof(float));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
	cuda_ret = hipMalloc((void**)&out2_d, num_elements*sizeof(float));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
	cuda_ret = hipMalloc((void**)&out3_d, SECTION_SIZE*sizeof(float));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
	cuda_ret = hipMalloc((void**)&out_d, num_elements*sizeof(float));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------
    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMemcpy(in_d, in_h, num_elements*sizeof(float),
        hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device");

	cuda_ret = hipMemset(out2_d, 0, num_elements*sizeof(float));
	if(cuda_ret != hipSuccess) FATAL("Unable to set device memory");

	cuda_ret = hipMemset(out3_d, 0, SECTION_SIZE*sizeof(float));
	if(cuda_ret != hipSuccess) FATAL("Unable to set device memory");

	cuda_ret = hipMemset(out_d, 0, num_elements*sizeof(float));
	if(cuda_ret != hipSuccess) FATAL("Unable to set device memory");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    //Set up and invoke your kernel inside the preScan function, which is in kernel.cu
    preScan(out2_d, in_d, num_elements);

	cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) FATAL("Unable to launch/execute kernel");

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel2 ----------------------------------------------------------
    printf("Launching kernel2..."); fflush(stdout);
    startTime(&timer);

    preScan2(out3_d, out2_d, num_elements);  // This time pass in out2_d (intermediate output) as in_d into preScan2

    cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) FATAL("Unable to launch/execute kernel");

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel3 ----------------------------------------------------------
    printf("Launching kernel3..."); fflush(stdout);
    startTime(&timer);

    preScan3(out_d, out3_d, out2_d, num_elements);  // This time pass in out2_d (intermediate output) as in_d into preScan2

    cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) FATAL("Unable to launch/execute kernel");

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy out_d from device to host ----------------------------------------

	printf("Copying final result from device to host..."); fflush(stdout);
	startTime(&timer);

	cuda_ret = hipMemcpy(out_h, out_d, num_elements*sizeof(float),
		hipMemcpyDeviceToHost);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");

	hipDeviceSynchronize();
	stopTime(&timer); printf("%f s\n", elapsedTime(timer));


    // Verify correctness -----------------------------------------------------

    printf("Verifying results...");

    verify(in_h, out_h, num_elements);

    // Free memory ------------------------------------------------------------

    hipFree(in_d); hipFree(out_d); hipFree(out2_d); hipFree(out3_d);
	free(in_h); free(out_h); free(out2_h); free(out3_h);

	return 0;
}

